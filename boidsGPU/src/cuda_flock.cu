#include "hip/hip_runtime.h"

#include "cuda_flock.cuh"

__global__ void updateBoidsKernel(float* boidPosX, float* boidPosY,
                                  float* boidVelX, float* boidVelY,
                                  float* accelerationX, float* accelerationY,
                                  int numBoids) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < numBoids) {
        // init constants
        float cohesion = 0.3;
        float alignment = 0.8;
        float separation = 3.0;
        float visionRadius = 5.0;
        float sqrVisionRadius = visionRadius * visionRadius;
        float maxVel = 1.0;
        float maxAccel = 0.1;
        // init arrays
        float cohesionForceX = 0.0, cohesionForceY = 0.0;
        float alignmentForceX = 0.0, alignmentForceY = 0.0;
        float separationForceX = 0.0, separationForceY = 0.0;
        float avgVelocityX = 0.0, avgVelocityY = 0.0;
        int neighbourCount = 0;
        float posX = boidPosX[i];
        float posY = boidPosY[i];
        float velX = boidVelX[i];
        float velY = boidVelY[i];
        for (int j = 0; j < numBoids; ++j) {
            float otherPosX = boidPosX[j];
            float otherPosY = boidPosY[j];
            
            float diffX = otherPosX - posX;
            float diffY = otherPosY - posY;
            float sqrDist = diffX * diffX + diffY * diffY;
            if (sqrDist < sqrVisionRadius) {
                cohesionForceX += diffX;
                cohesionForceY += diffY;
                avgVelocityX += boidVelX[j];
                avgVelocityY += boidVelY[j];

                if (sqrDist > 0) { 
                    float dist = sqrtf(sqrDist);
                    separationForceX += diffX / dist;
                    separationForceY += diffY / dist;
                }
                
                neighbourCount++;
            }
        }
        if (neighbourCount > 0) {
            // calculate cohesion
            cohesionForceX = (cohesionForceX / neighbourCount) * cohesion;
            cohesionForceY = (cohesionForceY / neighbourCount) * cohesion;

            // calculate alignment
            avgVelocityX /= neighbourCount;
            avgVelocityY /= neighbourCount;
            float avgVelMagnitude = sqrtf(avgVelocityX * avgVelocityX + avgVelocityY * avgVelocityY);
            if (avgVelMagnitude > 0) {
                float desiredVelX = (avgVelocityX / avgVelMagnitude) * maxVel;
                float desiredVelY = (avgVelocityY / avgVelMagnitude) * maxVel;
                alignmentForceX = (desiredVelX - velX) * alignment;
                alignmentForceY = (desiredVelY - velY) * alignment;
            }
            // calculate separation
            float sepMag = sqrtf(separationForceX * separationForceX + separationForceY * separationForceY);
            if (sepMag > 0) {
                separationForceX = (separationForceX / sepMag) * separation * -1.0;
                separationForceY = (separationForceY / sepMag) * separation * -1.0;
            }
            // limit forces
            float cohMag = sqrtf(cohesionForceX * cohesionForceX + cohesionForceY * cohesionForceY);
            if (cohMag > maxAccel) {
                cohesionForceX = (cohesionForceX / cohMag) * maxAccel;
                cohesionForceY = (cohesionForceY / cohMag) * maxAccel;
            }
            float alignMag = sqrtf(alignmentForceX * alignmentForceX + alignmentForceY * alignmentForceY);
            if (alignMag > maxAccel) {
                alignmentForceX = (alignmentForceX / alignMag) * maxAccel;
                alignmentForceY = (alignmentForceY / alignMag) * maxAccel;
            }
            
            if (sepMag > maxAccel) {
                separationForceX = (separationForceX / sepMag) * maxAccel;
                separationForceY = (separationForceY / sepMag) * maxAccel;
            }
            accelerationX[i] = cohesionForceX + alignmentForceX + separationForceX;
            accelerationY[i] = cohesionForceY + alignmentForceY + separationForceY;
        } else {
            accelerationX[i] = 0.0;
            accelerationY[i] = 0.0;
        }
    }
}
void updateBoidsCUDA(std::vector<float>& boidPosX, std::vector<float>& boidPosY,
                     std::vector<float>& boidVelX, std::vector<float>& boidVelY,
                     std::vector<float>& accelerationX, std::vector<float>& accelerationY) 
{
    int numBoids = boidPosX.size();
    // device pointers
    float *d_boidPosX, *d_boidPosY, *d_boidVelX, *d_boidVelY, *d_accelerationX, *d_accelerationY;
    // allocate device memory
    hipMalloc((void**)&d_boidPosX, numBoids * sizeof(float));
    hipMalloc((void**)&d_boidPosY, numBoids * sizeof(float));
    hipMalloc((void**)&d_boidVelX, numBoids * sizeof(float));
    hipMalloc((void**)&d_boidVelY, numBoids * sizeof(float));
    hipMalloc((void**)&d_accelerationX, numBoids * sizeof(float));
    hipMalloc((void**)&d_accelerationY, numBoids * sizeof(float));
    // cpu --> gpu
    hipMemcpy(d_boidPosX, boidPosX.data(), numBoids * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_boidPosY, boidPosY.data(), numBoids * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_boidVelX, boidVelX.data(), numBoids * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_boidVelY, boidVelY.data(), numBoids * sizeof(float), hipMemcpyHostToDevice);
    // run kernel func
    // 256 works fine
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBoids + threadsPerBlock - 1) / threadsPerBlock;
    updateBoidsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_boidPosX, d_boidPosY, 
                                                          d_boidVelX, d_boidVelY,
                                                          d_accelerationX, d_accelerationY, 
                                                          numBoids);

    // get acceleration updates back to cpu
    hipMemcpy(accelerationX.data(), d_accelerationX, numBoids * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(accelerationY.data(), d_accelerationY, numBoids * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_boidPosX);
    hipFree(d_boidPosY);
    hipFree(d_boidVelX);
    hipFree(d_boidVelY);
}
